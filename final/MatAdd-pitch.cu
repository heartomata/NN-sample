#include<stdio.h>
#include<hip/hip_runtime.h>


#define N 512
#define BLOCK_SIZE 16

__global__ void MatAdd(float *A, float *B, 
		       float *C, size_t pitch)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = j * N + i;
    if(i < N && j < N)
    {
	// compute C = A + B
    C[idx] = A[idx] + B[idx];

    }
}

int main()
{

    float h_A[N][N], h_B[N][N], h_C[N][N];
    float *d_A, *d_B, *d_C;

    size_t pitch;
    int i, j;

    // init data
    for(i = 0; i < N; i++)
    {
	for(j = 0; j < N; j++)
	{
	    h_A[i][j] = 1.0;
	    h_B[i][j] = 2.0;
	    h_C[i][j] = 0.0;
	}
    }

    // allocate device memory cudaMallocPitch
    hipMallocPitch((void**)&d_A, &pitch, N*sizeof(float), N);
    hipMallocPitch((void**)&d_B, &pitch, N*sizeof(float), N);
    hipMallocPitch((void**)&d_C, &pitch, N*sizeof(float), N);

    // transfer data to device cudaMemcpy2D
    hipMemcpy2D(d_A, pitch, h_A, N*sizeof(float), N*sizeof(float), N, hipMemcpyHostToDevice);
    hipMemcpy2D(d_B, pitch, h_B, N*sizeof(float), N*sizeof(float), N, hipMemcpyHostToDevice);
    hipMemcpy2D(d_C, pitch, h_C, N*sizeof(float), N*sizeof(float), N, hipMemcpyHostToDevice);

    // declare CTA
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlock(N/BLOCK_SIZE, N/BLOCK_SIZE);

    // MatAdd kernel
    MatAdd<<<numBlock, blockSize>>>(d_A, d_B, d_C, pitch);
    hipDeviceSynchronize();

    // transfer data back to host cudaMemcpy2D
    hipMemcpy2D(h_C, pitch, d_C, N*sizeof(float), N*sizeof(float), N, hipMemcpyDeviceToHost);

    // verify results
    int flag = 0;
    for(i = 0; i < N; i++)
    {
	for(j = 0; j < N; j++)
	{
	    if(h_C[i][j] != 3.0)
	    {
		flag = 1;
		printf("Error:%f, h[%d][%d]\n", h_C[i][j], i, j);
		break;
	    }
	}
    }

    if(!flag)
        printf("PASS\n");
    else
        printf("Fail\n");

    // free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
